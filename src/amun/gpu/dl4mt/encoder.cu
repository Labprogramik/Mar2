#include "hip/hip_runtime.h"
#include "encoder.h"
#include "common/sentences.h"

using namespace std;

namespace amunmt {
namespace GPU {

Encoder::Encoder(const Weights& model, const YAML::Node& config)
  : embeddings_(model.encEmbeddings_),
    forwardRnn_(InitForwardCell(model, config)),
    backwardRnn_(InitBackwardCell(model, config))
{}

std::unique_ptr<Cell> Encoder::InitForwardCell(const Weights& model, const YAML::Node& config){
  std::string celltype = config["enc-cell"] ? config["enc-cell"].as<std::string>() : "gru";
  if (celltype == "lstm") {
    return unique_ptr<Cell>(new LSTM<Weights::EncForwardLSTM>(*(model.encForwardLSTM_)));
  } else if (celltype == "mlstm") {
    return unique_ptr<Cell>(new Multiplicative<LSTM, Weights::EncForwardLSTM>(*model.encForwardMLSTM_));
  } else if (celltype == "gru") {
    return unique_ptr<Cell>(new GRU<Weights::EncForwardGRU>(*(model.encForwardGRU_)));
  }

  assert(false);
  return unique_ptr<Cell>(nullptr);
}

std::unique_ptr<Cell> Encoder::InitBackwardCell(const Weights& model, const YAML::Node& config){
  std::string enccell = config["enc-cell"] ? config["enc-cell"].as<std::string>() : "gru";
  std::string celltype = config["enc-cell-r"] ? config["enc-cell-r"].as<std::string>() : enccell;
  if (celltype == "lstm") {
    return unique_ptr<Cell>(new LSTM<Weights::EncBackwardLSTM>(*(model.encBackwardLSTM_)));
  } else if (celltype == "mlstm") {
    return unique_ptr<Cell>(new Multiplicative<LSTM, Weights::EncBackwardLSTM>(*model.encBackwardMLSTM_));
  } else if (celltype == "gru") {
    return unique_ptr<Cell>(new GRU<Weights::EncBackwardGRU>(*(model.encBackwardGRU_)));
  }

  assert(false);
  return unique_ptr<Cell>(nullptr);
}

size_t GetMaxLength(const Sentences& source, size_t tab) {
  size_t maxLength = source.at(0)->GetWords(tab).size();
  for (size_t i = 0; i < source.size(); ++i) {
    const Sentence &sentence = *source.at(i);
    maxLength = std::max(maxLength, sentence.GetWords(tab).size());
  }
  return maxLength;
}

std::vector<std::vector<FactWord>> GetBatchInput(const Sentences& source, size_t tab, size_t maxLen) {
  std::vector<std::vector<FactWord>> matrix(maxLen, std::vector<FactWord>(source.size()));

  for (size_t batchIdx = 0; batchIdx < source.size(); ++batchIdx) {
    for (size_t wordIdx = 0; wordIdx < source.at(batchIdx)->GetFactors(tab).size(); ++wordIdx) {
        matrix[wordIdx][batchIdx] = source.at(batchIdx)->GetFactors(tab)[wordIdx];
    }
  }

  return matrix;
}

void Encoder::Encode(const Sentences& source, size_t tab, mblas::Matrix& context,
                         mblas::IMatrix &sentenceLengths)
{
  size_t maxSentenceLength = GetMaxLength(source, tab);

  HostVector<uint> hSentenceLengths(source.size());
  for (size_t i = 0; i < source.size(); ++i) {
    hSentenceLengths[i] = source.at(i)->GetWords(tab).size();
  }

  sentenceLengths.NewSize(source.size(), 1, 1, 1);
  mblas::copy(thrust::raw_pointer_cast(hSentenceLengths.data()),
              hSentenceLengths.size(),
              sentenceLengths.data(),
              hipMemcpyHostToDevice);

  //cerr << "GetContext1=" << context.Debug(1) << endl;
  context.NewSize(maxSentenceLength,
                 forwardRnn_.GetStateLength().output + backwardRnn_.GetStateLength().output,
                 1,
                 source.size());
  //cerr << "GetContext2=" << context.Debug(1) << endl;

  auto input = GetBatchInput(source, tab, maxSentenceLength);

  for (size_t i = 0; i < input.size(); ++i) {
    if (i >= embeddedWords_.size()) {
      embeddedWords_.emplace_back();
    }
    embeddings_.Lookup(embeddedWords_[i], input[i]);
    //cerr << "embeddedWords_=" << embeddedWords_.back().Debug(true) << endl;
  }

  //cerr << "GetContext3=" << context.Debug(1) << endl;
  forwardRnn_.Encode(embeddedWords_.cbegin(),
                         embeddedWords_.cbegin() + maxSentenceLength,
                         context, source.size(), false);
  //cerr << "GetContext4=" << context.Debug(1) << endl;

  backwardRnn_.Encode(embeddedWords_.crend() - maxSentenceLength,
                          embeddedWords_.crend() ,
                          context, source.size(), true, &sentenceLengths);
  //cerr << "GetContext5=" << context.Debug(1) << endl;
}

}
}

